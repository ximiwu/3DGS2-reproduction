#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "backward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

__device__ __forceinline__ float sq(float x) { return x * x; }

__device__ inline float3 elementwiseMul(const glm::vec3& a, const glm::vec3& b) {
	return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}



__global__ void pos_solve_sh(){
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	// Compute intermediate values, as it is done during forward
	glm::vec3 pos = means[idx];
	glm::vec3 dir_orig = pos - campos;

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;

	// Use PyTorch rule for clamping: if clamping was applied,
	// gradient becomes 0.
	glm::vec3 dpixel_dRGB;
	dpixel_dRGB.x = clamped[3 * idx + 0] ? 0 : dpixel_dcolors[idx];
	dpixel_dRGB.y = clamped[3 * idx + 1] ? 0 : dpixel_dcolors[idx];
	dpixel_dRGB.z = clamped[3 * idx + 2] ? 0 : dpixel_dcolors[idx];


	x = dir_orig.x, y = dir_orig.y, z = dir_orig.z;
	xx = x * x, yy = y * y, zz = z * z;
	xy = x * y, yz = y * z, xz = x * z;

	float xyz = xy * z;
	float xxxx = xx * xx, yyyy = yy * yy, zzzz = zz * zz;
	float yyzz = yy * zz;


	float length = xx + yy + zz;
	float length1div2 = sqrtf(length);
	float length3div2 = length * length1div2;
	float length2 = length * length;
	float length5div2 = length2 * length1div2;
	float length3 = length2 * length;
	float length7div2 = length3 * length1div2;

	float inv_length3div2 = 1.0 / (length3div2 + 0.0000001f);
	float inv_length2 = 1.0 / (length2 + 0.0000001f);
	float inv_length5div2 = 1.0 / (length5div2 + 0.0000001f);
	float inv_length3 = 1.0 / (length3 + 0.0000001f);
	float inv_length7div2 = 1.0 / (length7div2 + 0.0000001f);


	glm::vec3 dRGBdx(0, 0, 0);
	glm::vec3 dRGBdy(0, 0, 0);
	glm::vec3 dRGBdz(0, 0, 0);


	if (deg > 0)
	{
		dRGBdx = (
			SH_C1 * sh[1] * xy
			-SH_C1 * sh[2] * xz
			-SH_C1 * sh[3] * (yy + zz)
		) * inv_length3div2;
		dRGBdy = (
			-SH_C1 * sh[1] * (xx + zz)
			-SH_C1 * sh[2] * yz
			+SH_C1 * sh[3] * xy
		) * inv_length3div2;
		dRGBdz = (
			SH_C1 * sh[1] * yz
			+SH_C1 * sh[2] * (xx + yy)
			+SH_C1 * sh[3] * xz
		) * inv_length3div2;
		if (deg > 1)
		{
			dRGBdx += (
				SH_C2[0] * sh[4] * y * (-xx + yy + zz)
				-SH_C2[1] * sh[5] * 2.0f * xyz
				-6.0f * SH_C2[2] * sh[6] * x * zz
				+SH_C2[3] * sh[7] * z * (-xx + yy + zz)
				+2.0f * SH_C2[4] * sh[8] * x * (2.0f * yy + zz)

			) * inv_length2;

			dRGBdy += (
				SH_C2[0] * sh[4] * x * (xx - yy + zz)
				+SH_C2[1] * sh[5] * z * (xx - yy + zz)
				-6.0f * SH_C2[2] * sh[6] * y * zz
				-2.0f * SH_C2[3] * sh[7] * xyz
				-2.0f * SH_C2[4] * sh[8] * y * (2.0f * xx + zz)

			) * inv_length2;

			dRGBdz += (
				-SH_C2[0] * sh[4] * 2.f * xyz
				+SH_C2[1] * sh[5] * y * (xx + yy - zz)
				+6.0f * SH_C2[2] * sh[6] * z * (xx + yy)
				+SH_C2[3] * sh[7] * x * (xx + yy - zz)
				+2.0f * SH_C2[4] * sh[8] * z * (yy - xx)

			) * inv_length2;


			if (deg > 2)
			{
				dRGBdx += (
					3.0f * SH_C3[0] * sh[9] * y * (-x*xx + 3.0f*x*yy + 2.0f*x*zz)
					+SH_C3[1] * sh[10] * yz * (-2.0f*xx + yy + zz)
					+SH_C3[2] * sh[11] * xy * (xx + yy - 14.0f*zz)
					+3.0f * SH_C3[3] * sh[12] * xz * ((xx + yy) - 4.0f * zz)
					-SH_C3[4] * sh[13] * (xx*(yy + 11.0f*zz) + yyyy - 3.0f*yyzz - 4.0f*zzzz)
					+SH_C3[5] * sh[14] * z * (-(x * xx) + 5.0f * x * yy + 2.0f * x * zz)
					+3.0f * SH_C3[6] * sh[15] * (xx * (3.0f * yy + zz) - yy * (yy + zz))

				) * inv_length5div2;

				dRGBdy += (
					3.0f * SH_C3[0] * sh[9] * (xxxx + xx*(zz - 3.0f*yy) - yyzz)
					+SH_C3[1] * sh[10] * xz * (xx - 2.0f*yy + zz)
					-SH_C3[2] * sh[11] * (xxxx + xx*(yy - 3.0f*zz) + 11.0f*yy*zz - 4.0f*zzzz)
					+3.0f * SH_C3[3] * sh[12] * yz * ((xx + yy) - 4.0f * zz)
					+SH_C3[4] * sh[13] * xy * (xx + yy - 14.0f*zz)
					+SH_C3[5] * sh[14] * yz * (-5.0f * xx + yy - 2.0f * zz)
					-3.0f * SH_C3[6] * sh[15] * xy * (3.0f * xx - yy + 2.0f * zz)

				) * inv_length5div2;


				dRGBdz += (
					3.0f * SH_C3[0] * sh[9] * yz * (yy - 3.0f*xx)
					+SH_C3[1] * sh[10] * xy * (xx + yy - 2.0f*zz)
					+SH_C3[2] * sh[11] * yz * (11.0f*xx + 11.0f*yy - 4.0f*zz)
					-3.0f * SH_C3[3] * sh[12] * (xx + yy) * ((xx + yy) - 4.0f * zz)
					+SH_C3[4] * sh[13] * xz * (11.0f*xx + 11.0f*yy - 4.0f*zz)
					+SH_C3[5] * sh[14] * (xx - yy) * (xx + yy - 2.0f * zz)
					-3.0f * SH_C3[6] * sh[15] * xz * (xx - 3.0f * yy)

				) * inv_length5div2;
			}
		}
	}
	
	float3 dpixel_dmean_x = elementwiseMul(dRGBdx, dpixel_dRGB);
	float3 dpixel_dmean_y = elementwiseMul(dRGBdy, dpixel_dRGB);
	float3 dpixel_dmean_z = elementwiseMul(dRGBdz, dpixel_dRGB);


	//here,xyz denote means3D
	glm::vec3 dRGBdxx(0, 0, 0);
	glm::vec3 dRGBdxy(0, 0, 0);
	glm::vec3 dRGBdxz(0, 0, 0);
	glm::vec3 dRGBdyy(0, 0, 0);
	glm::vec3 dRGBdyz(0, 0, 0);
	glm::vec3 dRGBdzz(0, 0, 0);

	if (deg > 0)
	{
		dRGBdxx = (
			SH_C1 * sh[1] * y * (-2.f * xx + yy + zz)
			-SH_C1 * sh[2] * z * (-2.f * xx + yy + zz)
			+3.f * SH_C1 * sh[3] * x * (yy + zz)

		) * inv_length5div2;
		
		dRGBdxy = (
			SH_C1 * sh[1] * x * (xx - 2.f * yy + zz)
			+3.f * SH_C1 * sh[2] * xyz
			+SH_C1 * sh[3] * y * (-2.f * xx + yy + zz)

		) * inv_length5div2;
		
		dRGBdxz = (
			-3.f * SH_C1 * sh[1] * xyz
			-SH_C1 * sh[2] * x * (xx + yy - 2.f * zz)
			+SH_C1 * sh[3] * z * (-2.f * xx + yy + zz)

		) * inv_length5div2;
		
		
		dRGBdyy = (
			3.f * SH_C1 * sh[1] * y * (xx + zz)
			-SH_C1 * sh[2] * z * (xx - 2.f * yy + zz)
			+SH_C1 * sh[3] * x * (xx - 2.f * yy + zz)

		) * inv_length5div2;
		
		dRGBdyz = (
			SH_C1 * sh[1] * z * (xx - 2.f * yy + zz)
			-SH_C1 * sh[2] * y * (xx + yy - 2.f * zz)
			-3.f * SH_C1 * sh[3] * xyz

		) * inv_length5div2;
		
		dRGBdzz = (
			SH_C1 * sh[1] * y * (xx + yy - 2.f * zz)
			-3.f * SH_C1 * sh[2] * z * (xx + yy)
			+SH_C1 * sh[3] * x * (xx + yy - 2.f * zz)

		) * inv_length5div2;

		if (deg > 1)
		{
			dRGBdxx += (
				SH_C2[0] * sh[4] * 2.f * xy * (xx - 3.f * (yy + zz))
				-SH_C2[1] * sh[5] * 2.f * yz * (-3.f * xx + yy + zz)
				-SH_C2[2] * sh[6] * 6.f * zz * (-3.f * xx + yy + zz)
				+SH_C2[3] * sh[7] * 2.f * xz * (xx - 3.f * (yy + zz))
				-SH_C2[4] * sh[8] * 2.f * (2.f*yy + zz) * (3.f*xx - yy - zz)

			) * inv_length3;
			dRGBdxy += (
				-SH_C2[0] * sh[4] * (xxxx - 6.f * xx * yy + yyyy - zzzz)
				-SH_C2[1] * sh[5] * 2.f * xz * (xx - 3.f * yy + zz)
				+SH_C2[2] * sh[6] * 24.f * xy * zz
				-SH_C2[3] * sh[7] * 2.f * yz * (-3.f * xx + yy + zz)
				+SH_C2[4] * sh[8] * 8.f * xy * (xx - yy)

			) * inv_length3;
			dRGBdxz += (
				-SH_C2[0] * sh[4] * 2.f * yz * (-3.f * xx + yy + zz)
				-SH_C2[1] * sh[5] * 2.f * xy * (xx + yy - 3.f * zz)
				-SH_C2[2] * sh[6] * 12.f * xz * (xx + yy - zz)
				-SH_C2[3] * sh[7] * (xxxx - 6.f * xx * zz - yyyy + zzzz)
				+SH_C2[4] * sh[8] * 4.f * xz * (xx - 3.f*yy - zz)

			) * inv_length3;
			dRGBdyy += (
				-SH_C2[0] * sh[4] * 2.f * xy * (3.f * xx - yy + 3.f * zz)
				+SH_C2[1] * sh[5] * 2.f * yz * (-3.f * xx + yy - 3.f * zz)
				-SH_C2[2] * sh[6] * 6.f * zz * (xx - 3.f * yy + zz)
				-SH_C2[3] * sh[7] * 2.f * xz * (xx - 3.f * yy + zz)
				-SH_C2[4] * sh[8] * 2.f * (2.f*xx + zz) * (xx - 3.f*yy + zz)

			) * inv_length3;
			dRGBdyz += (
				-SH_C2[0] * sh[4] * 2.f * xz * (xx - 3.f * yy + zz)
				+SH_C2[1] * sh[5] * (xxxx - yyyy + 6.f * yyzz - zzzz)
				-SH_C2[2] * sh[6] * 12.f * yz * (xx + yy - zz)
				-SH_C2[3] * sh[7] * 2.f * xy * (xx + yy - 3.f * zz)
				+SH_C2[4] * sh[8] * 4.f * yz * (3.f*xx - yy + zz)

			) * inv_length3;
			dRGBdzz += (
				-SH_C2[0] * sh[4] * 2.f * xy * (xx + yy - 3.f * zz)
				-SH_C2[1] * sh[5] * 2.f * yz * (3.f * xx + 3.f * yy - zz)
				+SH_C2[2] * sh[6] * 6.f * (xx + yy) * (xx + yy - 3.f * zz)
				-SH_C2[3] * sh[7] * 2.f * xz * (3.f * xx + 3.f * yy - zz)
				-SH_C2[4] * sh[8] * 2.f * (xx - yy) * (xx + yy - 3.f*zz)

			) * inv_length3;

			if (deg > 2)
			{

				dRGBdxx += (
					3.f * SH_C3[0] * sh[9] * y * (2.f * xxxx - xx * (15.f * yy + 11.f * zz) + 3.f * yyyy + 5.f * yyzz + 2.f * zzzz)
					-3.f * SH_C3[1] * sh[10] * yz * (3.f * x * (yy + zz) - 2.f * x * xx)
					+SH_C3[2] * sh[11] * y * (-2.f * xxxx - xx * (yy - 59.f * zz) + yyyy - 13.f * yyzz - 14.f * zzzz)
					-3.f * SH_C3[3] * sh[12] * z * (2.f * xxxx + xx * (yy - 19.f * zz) - yyyy + 3.f * yyzz + 4.f * zzzz)
					+3.f * SH_C3[4] * sh[13] * x * (xx * (yy + 11.f * zz) + yyyy - 13.f * yyzz - 14.f * zzzz)
					+SH_C3[5] * sh[14] * z * (2.f * xxxx - xx * (23.f * yy + 11.f * zz) + 5.f * yyyy + 7.f * yyzz + 2.f * zzzz)
					-3.f * SH_C3[6] * sh[15] * x * (3.f * xx * (3.f * yy + zz) - 11.f * yyyy - 13.f * yyzz - 2.f * zzzz)

				) * inv_length7div2;
		
				dRGBdxy += (
					-3.f * SH_C3[0] * sh[9] * x * (xxxx - xx * (13.f * yy + zz) + 6.f * yyyy - yyzz - 2.f * zzzz)
					+SH_C3[1] * sh[10] * z * (-2.f * xxxx + xx * (11.f * yy - zz) - 2.f * yyyy - yyzz + zzzz)
					+SH_C3[2] * sh[11] * x * (xxxx - xx * (yy + 13.f * zz) - 2.f * yyyy + 59.f * yyzz - 14.f * zzzz)
					-3.f * SH_C3[3] * sh[12] * xyz * (3.f * xx + 3.f * yy - 22.f * zz)
					-SH_C3[4] * sh[13] * y * (2 * xxxx + xx * (yy - 59.f * zz) - yyyy + 13.f * yyzz + 14.f * zzzz)
					+15.f * SH_C3[5] * sh[14] * xyz * (xx - yy)
					+3.f * SH_C3[6] * sh[15] * y * (6.f * xxxx - xx * (13.f * yy + zz) + yyyy - yyzz - 2.f * zzzz)

				) * inv_length7div2;
		
				dRGBdxz += (
					3.f * SH_C3[0] * sh[9] * xyz * (9.f * xx - 11.f * yy - 6.f * zz)
					+SH_C3[1] * sh[10] * y * (-2.f * xxxx - xx * (yy - 11.f * zz) + yyyy - yyzz - 2.f * zzzz)
					-3.f * SH_C3[2] * sh[11] * xyz * (11.f * xx + 11.f * yy - 14.f * zz)
					+3.f * SH_C3[3] * sh[12] * x * (xxxx + 2.f * xx * (yy - 8.f * zz) + yyyy - 16.f * yyzz + 8.f * zzzz)
					-SH_C3[4] * sh[13] * z * (22.f * xxxx + xx * (11.f * yy - 49.f * zz) - 11.f * yyyy - 7.f * yyzz + 4.f * zzzz)
					-SH_C3[5] * sh[14] * x * (xxxx - 2.f * xx * (2.f * yy + 5 * zz) - 5.f * yyyy + 14.f * yyzz + 4.f * zzzz)
					+3.f * SH_C3[6] * sh[15] * z * (2.f * xxxx - 3.f * xx * (5.f * yy + zz) + 3.f * yy * (yy + zz))

				) * inv_length7div2;
		
		
				dRGBdyy += (
					-3.f * SH_C3[0] * sh[9] * y * (11.f * xxxx + xx * (13.f * zz - 9.f * yy) - 3.f * yyzz + 2.f * zzzz)
					-3.f * SH_C3[1] * sh[10] * xyz * (3.f * xx - 2.f * yy + 3.f * zz)
					+3.f * SH_C3[2] * sh[11] * y * (xxxx + xx * (yy - 13.f * zz) + 11.f * yyzz - 14.f * zzzz)
					-3.f * SH_C3[3] * sh[12] * z * (-xxxx + xx * (yy + 3.f * zz) + 2.f * yyyy - 19.f * yyzz + 4.f * zzzz)
					+SH_C3[4] * sh[13] * x * (xxxx - xx * (yy + 13.f * zz) - 2.f * yyyy + 59.f * yyzz - 14.f * zzzz)
					-SH_C3[5] * sh[14] * z * (5.f * xxxx + xx * (7.f * zz - 23.f * yy) + 2.f * yyyy - 11.f * yyzz + 2.f * zzzz)
					-3.f * SH_C3[6] * sh[15] * x * (3.f * xxxx + 5.f * xx * (zz - 3.f * yy) + 2.f * yyyy - 11.f * yyzz + 2.f * zzzz)

				) * inv_length7div2;
		
				dRGBdyz += (
					-3.f * SH_C3[0] * sh[9] * z * (3.f * xxxx + 3.f * xx * (zz - 5.f * yy) + 2.f * yyyy - 3.f * yyzz)
					+SH_C3[1] * sh[10] * x * (xxxx - xx * (yy + zz) - 2.f * yyyy + 11.f * yyzz - 2.f * zzzz)
					+SH_C3[2] * sh[11] * z * (11.f * xxxx + xx * (7.f * zz - 11.f * yy) - 22.f * yyyy + 49.f * yyzz - 4.f * zzzz)
					+3.f * SH_C3[3] * sh[12] * y * (xxxx + 2.f * xx * (yy - 8.f * zz) + yyyy - 16.f * yyzz + 8.f * zzzz)
					-3.f * SH_C3[4] * sh[13] * xyz * (11.f * xx + 11.f * yy - 14.f * zz)
					+SH_C3[5] * sh[14] * y * (-5.f * xxxx + xx * (14.f * zz - 4.f * yy) + yyyy - 10.f * yyzz + 4.f * zzzz)
					+3.f * SH_C3[6] * sh[15] * xyz * (11.f * xx - 9.f * yy + 6.f * zz)

				) * inv_length7div2;
		
				dRGBdzz += (
					3.f * SH_C3[0] * sh[9] * y * (yy - 3.f * xx) * (xx + yy - 4.f * zz)
					-3.f * SH_C3[1] * sh[10] * xyz * (3.f * xx + 3.f * yy - 2.f * zz)
					+SH_C3[2] * sh[11] * y * (11.f * xxxx + xx * (22.f * yy - 56.f * zz) + 11.f * yyyy - 56.f * yyzz + 8.f * zzzz)
					+3.f * SH_C3[3] * sh[12] * z * (xx + yy) * (13.f * xx + 13.f * yy - 12.f * zz)
					+SH_C3[4] * sh[13] * x * (11 * xxxx + xx * (22.f * yy - 56.f * zz) + 11.f * yyyy - 56.f * yyzz + 8.f * zzzz)
					-3.f * SH_C3[5] * sh[14] * z * (xx - yy) * (3.f * xx + 3.f * yy - 2.f * zz)
					-3.f * SH_C3[6] * sh[15] * x * (xx - 3.f * yy) * (xx + yy - 4.f * zz)

				) * inv_length7div2;
			}
		}
	}


	// [a,b,c]
	// [d,e,f]
	// [g,h,i]
	float3 dpixel_dmean_2_a = elementwiseMul(dRGBdxx, dpixel_dRGB);
	float3 dpixel_dmean_2_b = elementwiseMul(dRGBdxy, dpixel_dRGB);
	float3 dpixel_dmean_2_c = elementwiseMul(dRGBdxz, dpixel_dRGB);
	float3 dpixel_dmean_2_e = elementwiseMul(dRGBdyy, dpixel_dRGB);
	float3 dpixel_dmean_2_f = elementwiseMul(dRGBdyz, dpixel_dRGB);
	float3 dpixel_dmean_2_i = elementwiseMul(dRGBdzz, dpixel_dRGB);



}

__global__ void pos_solve_proj() {
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	float3 m = means[idx];

	// Taking care of gradients from the screenspace points
	float4 m_clip = transformPoint4x4(m, proj);
	float inv_m_clip_w = 1.0f / (m_clip.w + 0.0000001f);
	float inv_m_clip_w_2 = inv_m_clip_w * inv_m_clip_w;


	// Compute loss gradient w.r.t. 3D means due to gradients of 2D means
	// from rendering procedure
	float3 dG_dmean;
	float mul1 = (proj[0] * m.x + proj[4] * m.y + proj[8] * m.z + proj[12]);
	float mul2 = (proj[1] * m.x + proj[5] * m.y + proj[9] * m.z + proj[13]);
	float mul1_2 = mul1 * inv_m_clip_w_2;
	float mul2_2 = mul2 * inv_m_clip_w_2;
	float mul1_3 = mul1_2 * inv_m_clip_w;
	float mul2_3 = mul2_2 * inv_m_clip_w;


	// [a,b,c]
	// [d,e,f]
	float2x3 dmean2D_dmean3D;

	dmean2D_dmean3D[0].x = proj[0] * inv_m_clip_w - proj[3] * mul1_2;
	dmean2D_dmean3D[0].y = proj[4] * inv_m_clip_w - proj[7] * mul1_2;
	dmean2D_dmean3D[0].z = proj[8] * inv_m_clip_w - proj[11] * mul1_2;

	dmean2D_dmean3D[1].x = proj[1] * inv_m_clip_w - proj[3] * mul2_2;
	dmean2D_dmean3D[1].y = proj[5] * inv_m_clip_w - proj[7] * mul2_2;
	dmean2D_dmean3D[1].z = proj[9] * inv_m_clip_w - proj[11] * mul2_2;



	dG_dmean.x = (dmean2D_dmean3D[0].x) * dG_dmean2D[idx].x + (dmean2D_dmean3D[1].x) * dG_dmean2D[idx].y;
	dG_dmean.y = (dmean2D_dmean3D[0].y) * dG_dmean2D[idx].x + (dmean2D_dmean3D[1].y) * dG_dmean2D[idx].y;
	dG_dmean.z = (dmean2D_dmean3D[0].z) * dG_dmean2D[idx].x + (dmean2D_dmean3D[1].z) * dG_dmean2D[idx].y;


	// [a,b,c]
	// [d,e,f]
	// [g,h,i]

	float2x3 dG_dmean_2;


	dG_dmean_2[0].x = dG_dmean2D[idx].x * (-2.f * proj[0] * proj[3] * inv_m_clip_w_2 + 2.f * proj[3] * proj[3] * mul1_3) + dG_dmean2D[idx].y * (-2.f * proj[1] * proj[3] * inv_m_clip_w_2 + 2.f * proj[3] * proj[3] * mul2_3);
	dG_dmean_2[0].y = dG_dmean2D[idx].x * (2.f * proj[3] * proj[7] * mul1_3 + (- proj[0] * proj[7] - proj[3] * proj[4]) * inv_m_clip_w_2) + dG_dmean2D[idx].y * (2.f * proj[3] * proj[7] * mul2_3 + (- proj[1] * proj[7] - proj[3] * proj[5]) * inv_m_clip_w_2);
	dG_dmean_2[0].z = dG_dmean2D[idx].x * (2.f * proj[11] * proj[3] * mul1_3 - (proj[0] * proj[11] + proj[3] * proj[8]) * inv_m_clip_w_2) + dG_dmean2D[idx].y * (2.f * proj[11] * proj[3] * mul2_3 - (proj[1] * proj[11] + proj[3] * proj[9]) * inv_m_clip_w_2);
	dG_dmean_2[1].x = dG_dmean2D[idx].x * (2.f * proj[7] * proj[7] * mul1_3 - 2.f * proj[4] * proj[7] * inv_m_clip_w_2) + dG_dmean2D[idx].y * (2.f * proj[7] * proj[7] * mul2_3 - 2.f * proj[5] * proj[7] * inv_m_clip_w_2);
	dG_dmean_2[1].y = dG_dmean2D[idx].x * (2.f * proj[11] * proj[7] * mul1_3 - (proj[11] * proj[4] + proj[7] * proj[8]) * inv_m_clip_w_2) + dG_dmean2D[idx].y * (2.f * proj[11] * proj[7] * mul2_3 - (proj[11] * proj[5] + proj[7] * proj[9]) * inv_m_clip_w_2);
	dG_dmean_2[1].z = dG_dmean2D[idx].x * (2.f * proj[11] * (proj[11] * mul1_3 - proj[8] * inv_m_clip_w_2)) + dG_dmean2D[idx].y * (2.f * proj[11] * (proj[11] * mul2_3 - proj[9] * inv_m_clip_w_2));

	float mul3_1_1 = dmean2D_dmean3D[0].x * dG_dmean2D_2.x + dmean2D_dmean3D[1].x * dG_dmean2D_2.y;
	float mul3_1_2 = dmean2D_dmean3D[0].x * dG_dmean2D_2.y + dmean2D_dmean3D[1].x * dG_dmean2D_2.z;
	float mul3_2_1 = dmean2D_dmean3D[0].y * dG_dmean2D_2.x + dmean2D_dmean3D[1].y * dG_dmean2D_2.y;
	float mul3_2_2 = dmean2D_dmean3D[0].y * dG_dmean2D_2.y + dmean2D_dmean3D[1].y * dG_dmean2D_2.z;
	float mul3_3_1 = dmean2D_dmean3D[0].z * dG_dmean2D_2.x + dmean2D_dmean3D[1].z * dG_dmean2D_2.y;
	float mul3_3_2 = dmean2D_dmean3D[0].z * dG_dmean2D_2.y + dmean2D_dmean3D[1].z * dG_dmean2D_2.z;

	dG_dmean_2[0].x += dmean2D_dmean3D[0].x * mul3_1_1 + dmean2D_dmean3D[1].x * mul3_1_2;
	dG_dmean_2[0].y += dmean2D_dmean3D[0].y * mul3_1_1 + dmean2D_dmean3D[1].y * mul3_1_2;
	dG_dmean_2[0].z += dmean2D_dmean3D[0].z * mul3_1_1 + dmean2D_dmean3D[1].z * mul3_1_2;
	dG_dmean_2[1].x += dmean2D_dmean3D[0].y * mul3_2_1 + dmean2D_dmean3D[1].y * mul3_2_2;
	dG_dmean_2[1].y += dmean2D_dmean3D[0].z * mul3_2_1 + dmean2D_dmean3D[1].z * mul3_2_2;
	dG_dmean_2[1].z += dmean2D_dmean3D[0].z * mul3_3_1 + dmean2D_dmean3D[1].z * mul3_3_2;









	const float* cov3D = cov3Ds + 6 * idx;

	float3 mean = means[idx];
	float3 t = transformPoint4x3(mean, view_matrix);
	

	const float t_z_inv = 1.0f / t.z;
	const float t_z_2_inv = 1.0f / (t.z * t.z);
	const float t_z_3_inv = 1.0f / (t_z_inv * t_z_2_inv);
	const float t_z_4_inv = 1.0f / (t_z_2_inv * t_z_2_inv);

	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x * t_z_inv;
	const float tytz = t.y * t_z_inv;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;
	
	const float x_grad_mul = txtz < -limx || txtz > limx ? 0 : 1;
	const float y_grad_mul = tytz < -limy || tytz > limy ? 0 : 1;


	glm::mat3 J = glm::mat3(h_x * t_z_inv, 0.0f, 0.0f,
		0.0f, h_y * t_z_inv, 0.0f,
		-(h_x * t.x) * t_z_2_inv, -(h_y * t.y) * t_z_2_inv, 0.0f);


	glm::mat3 W = glm::mat3(
		view_matrix[0], view_matrix[1], view_matrix[2],
		view_matrix[4], view_matrix[5], view_matrix[6],
		view_matrix[8], view_matrix[9], view_matrix[10]);

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);



	glm::mat3 term1 = W * Vrk * glm::transpose(W);
	glm::mat3 term2 = term1 * glm::transpose(J);
	glm::mat3 term3 = J * term1;

	glm::mat3 cov2D = J * term2;

	float c_xx = cov2D[0].x;
	float c_xy = cov2D[0].y;
	float c_yy = cov2D[1].y;
	
	constexpr float h_var = 0.3f;

	c_xx += h_var;
	c_yy += h_var;


	const float P_c_xy_2 = c_xy * c_xy;
	const float P_c_xy_3 = P_c_xy_2 * c_xy;
	const float P_c_xy_4 = P_c_xy_2 * P_c_xy_2;

	const float P_c_xx_2 = c_xx * c_xx;
	const float P_c_xx_3 = P_c_xx_2 * c_xx;
	
	const float P_c_yy_2 = c_yy * c_yy;
	const float P_c_yy_3 = P_c_yy_2 * c_yy;

	const float P_c_xx_xy = c_xx * c_xy;
	const float P_c_xx_yy = c_xx * c_yy;
	const float P_c_xy_yy = c_xy * c_yy;
	const float P_c_xx_xy_yy = P_c_xx_xy * c_yy;

	float denom = P_c_xy_2 - P_c_xx_yy;

	float denom2inv = 1.0f / ((denom * denom) + 0.0000001f);
	float denom3inv = 1.0f / ((denom * denom * denom) + 0.0000001f);

	float3x3 dconic2D_dcov2D;

	dconic2D_dcov2D[0].x = -P_c_yy_2 * denom2inv;
	dconic2D_dcov2D[0].y = P_c_xy_yy * denom2inv;
	dconic2D_dcov2D[0].z = -P_c_xy_2 * denom2inv;

	dconic2D_dcov2D[1].x = dconic2D_dcov2D[0].y;
	//(in cuda) dconic2D_dcov2D[1].y = (in math) dconic2D_dcov2D[1,2,1,2] + dconic2D_dcov2D[1,2,2,1] = (in math) dconic2D_dcov2D[2,1,1,2] + dconic2D_dcov2D[2,1,2,1]
	dconic2D_dcov2D[1].y = -(P_c_xx_yy + P_c_xy_2) * denom2inv;
	dconic2D_dcov2D[1].z = P_c_xx_xy * denom2inv;

	dconic2D_dcov2D[2].x = dconic2D_dcov2D[0].z;
	dconic2D_dcov2D[2].y = dconic2D_dcov2D[1].z;
	dconic2D_dcov2D[2].z = -P_c_xx_2 * denom2inv;



	float dG_dc_xx = 0;
	float dG_dc_xy = 0;
	float dG_dc_yy = 0;


	if (denom2inv != 0)
	{
		dG_dc_xx = denom2inv * (-P_c_xy_2 * dG_dconic2D.x + 2 * P_c_xy_yy * dG_dconic2D.y + (- P_c_xy_2) * dG_dconic2D.z);
		dG_dc_xy = denom2inv * (P_c_xy_yy * dG_dconic2D.x - (denom + 2 * P_c_xy_2) * dG_dconic2D.y + P_c_xx_xy * dG_dconic2D.z);
		dG_dc_yy = denom2inv * (-P_c_xy_2 * dG_dconic2D.z + 2 * P_c_xx_xy * dG_dconic2D.y + (- P_c_xy_2) * dG_dconic2D.x);
	}





	float3x3 dG_dcov2D_2;
	dG_dcov2D_2[0].x = denom3inv * (
		2.0f * P_c_xy_3 * c_yy * dG_dconic2D_dcov2D[1].x 
		- P_c_xy_4 * dG_dconic2D_dcov2D[2].x 
		+ P_c_yy_3 * (c_xx * dG_dconic2D_dcov2D[0].x - 2.0f * dG_dconic2D.x) 
		- 2.0f * c_xy * P_c_yy_2 * (c_xx * dG_dconic2D_dcov2D[1].x - 2.0f * dG_dconic2D.y) 
		+ P_c_xy_2 * c_yy * (-c_yy * dG_dconic2D_dcov2D[0].x + c_xx * dG_dconic2D_dcov2D[2].x - 2.0f * dG_dconic2D.z)
	);
	dG_dcov2D_2[0].y = denom3inv * (
		P_c_xy_2 * c_yy * (c_xx * dG_dconic2D_dcov2D[2].y - c_yy * dG_dconic2D_dcov2D[0].y - 3.0f * dG_dconic2D.y) +
		P_c_xy_yy * (c_xx * (dG_dconic2D.z - 2.0f * c_yy * dG_dconic2D_dcov2D[1].y) + 2.0f * c_yy * dG_dconic2D.x) +
		c_xx * P_c_yy_2 * (c_yy * dG_dconic2D_dcov2D[0].y - dG_dconic2D.y) +
		(-P_c_xy_4 * dG_dconic2D_dcov2D[2].y) +
		P_c_xy_3 * (2.0f * c_yy * dG_dconic2D_dcov2D[1].y + dG_dconic2D.z)
	);
	dG_dcov2D_2[0].z = denom3inv * (
		-P_c_xy_2 * (-P_c_xx_yy * dG_dconic2D_dcov2D[2].z + 2.0f * c_xx * dG_dconic2D.z + P_c_yy_2 * dG_dconic2D_dcov2D[0].z + 2.0f * c_yy * dG_dconic2D.x) +
		2.0f * P_c_xx_xy_yy * (dG_dconic2D.y - c_yy * dG_dconic2D_dcov2D[1].z) +
		c_xx * P_c_yy_3 * dG_dconic2D_dcov2D[0].z +
		(-P_c_xy_4 * dG_dconic2D_dcov2D[2].z) +
		2.0f * P_c_xy_3 * (c_yy * dG_dconic2D_dcov2D[1].z + dG_dconic2D.y)
	);

	dG_dcov2D_2[1].x = denom3inv * (
		P_c_xy_yy * (-P_c_xx_2 * dG_dconic2D_dcov2D[2].x + c_xx * (dG_dconic2D.z - c_yy * dG_dconic2D_dcov2D[0].x) + 2.0f * c_yy * dG_dconic2D.x) +
		P_c_xy_3 * (c_xx * dG_dconic2D_dcov2D[2].x + c_yy * dG_dconic2D_dcov2D[0].x + dG_dconic2D.z) +
		c_xx * P_c_yy_2 * (c_xx * dG_dconic2D_dcov2D[1].x - dG_dconic2D.y) +
		(-P_c_xy_4 * dG_dconic2D_dcov2D[1].x) +
		(-3.0f * P_c_xy_2 * c_yy * dG_dconic2D.y)
	);
	//(in cuda) dG_dcov2D_2[1].y = (in math) dG_dcov2D_2[1,2,1,2]+dG_dcov2D_2[1,2,2,1] = (in math) dG_dcov2D_2[2,1,1,2]+dG_dcov2D_2[2,1,2,1]
	dG_dcov2D_2[1].y = 
		denom3inv * (
		P_c_xx_2 * P_c_yy_2 * dG_dconic2D_dcov2D[1].y +
		P_c_xy_3 * (c_xx * dG_dconic2D_dcov2D[2].y + c_yy * dG_dconic2D_dcov2D[0].y + 2.0f * dG_dconic2D.y) +
		(-2.0f * P_c_xy_2 * (c_xx * dG_dconic2D.z + c_yy * dG_dconic2D.x)) +
		(-P_c_xx_xy_yy * (c_xx * dG_dconic2D_dcov2D[2].y + c_yy * dG_dconic2D_dcov2D[0].y - 2.0f * dG_dconic2D.y)) +
		(-P_c_xy_4 * dG_dconic2D_dcov2D[1].y)
		+ denom3inv * (
			P_c_xy_3 * (c_xx * dG_dconic2D_dcov2D[2].y + c_yy * dG_dconic2D_dcov2D[0].y) +
			(-P_c_xy_2 * (c_xx * dG_dconic2D.z + c_yy * dG_dconic2D.x)) +
			(-P_c_xx_xy_yy * (c_xx * dG_dconic2D_dcov2D[2].y + c_yy * dG_dconic2D_dcov2D[0].y - 4.0f * dG_dconic2D.y)) +
			P_c_xx_yy * (P_c_xx_yy * dG_dconic2D_dcov2D[1].y - c_xx * dG_dconic2D.z - c_yy * dG_dconic2D.x) +
			(-P_c_xy_4 * dG_dconic2D_dcov2D[1].y)
		);
	);
	dG_dcov2D_2[1].z = denom3inv * (
		P_c_xx_2 * c_yy * (c_yy * dG_dconic2D_dcov2D[1].z - dG_dconic2D.y) +
		P_c_xy_3 * (c_xx * dG_dconic2D_dcov2D[2].z + c_yy * dG_dconic2D_dcov2D[0].z + dG_dconic2D.x) +
		(-3.0f * c_xx * P_c_xy_2 * dG_dconic2D.y) +
		P_c_xx_xy * (c_yy * (dG_dconic2D.x - c_xx * dG_dconic2D_dcov2D[2].z) + 2.0f * c_xx * dG_dconic2D.z - P_c_yy_2 * dG_dconic2D_dcov2D[0].z) +
		(-P_c_xy_4 * dG_dconic2D_dcov2D[1].z)
	);

	dG_dcov2D_2[2].x = denom3inv * (
		(c_xx * c_xx * c_xx) * c_yy * dG_dconic2D_dcov2D[2].x +
		(-P_c_xy_2 * (P_c_xx_2 * dG_dconic2D_dcov2D[2].x - P_c_xx_yy * dG_dconic2D_dcov2D[0].x + 2.0f * c_xx * dG_dconic2D.z + 2.0f * c_yy * dG_dconic2D.x)) +
		2.0f * P_c_xy_3 * (c_xx * dG_dconic2D_dcov2D[1].x + dG_dconic2D.y) +
		2.0f * P_c_xx_xy_yy * (dG_dconic2D.y - c_xx * dG_dconic2D_dcov2D[1].x) +
		(-P_c_xy_4 * dG_dconic2D_dcov2D[0].x)
	);
	dG_dcov2D_2[2].y = denom3inv * (
		P_c_xx_2 * c_yy * (c_xx * dG_dconic2D_dcov2D[2].y - dG_dconic2D.y) +
		P_c_xy_3 * (2.0f * c_xx * dG_dconic2D_dcov2D[1].y + dG_dconic2D.x) +
		c_xx * P_c_xy_2 * (-(c_xx) * dG_dconic2D_dcov2D[2].y + c_yy * dG_dconic2D_dcov2D[0].y - 3.0f * dG_dconic2D.y) +
		P_c_xx_xy * (-2.0f * P_c_xx_yy * dG_dconic2D_dcov2D[1].y + 2.0f * c_xx * dG_dconic2D.z + c_yy * dG_dconic2D.x) +
		(-P_c_xy_4 * dG_dconic2D_dcov2D[0].y)
	);
	dG_dcov2D_2[2].z = denom3inv * (
		(c_xx * c_xx * c_xx) * (c_yy * dG_dconic2D_dcov2D[2].z - 2.0f * dG_dconic2D.z) +
		(-2.0f * P_c_xx_2 * c_xy * (c_yy * dG_dconic2D_dcov2D[1].z - 2.0f * dG_dconic2D.y)) +
		2.0f * c_xx * P_c_xy_3 * dG_dconic2D_dcov2D[1].z +
		c_xx * P_c_xy_2 * (-(c_xx) * dG_dconic2D_dcov2D[2].z + c_yy * dG_dconic2D_dcov2D[0].z - 2.0f * dG_dconic2D.x) +
		(-P_c_xy_4 * dG_dconic2D_dcov2D[0].z)
	);






	float4x3 dJ_dmean;
	

	const float h_x_div_t_z_2 = J[0].x * t_z_inv;
	const float h_y_div_t_z_2 = J[1].y * t_z_inv;
	const float h_x_t_x_div_t_z_3 = -J[2].x * t_z_inv;
	const float h_y_t_y_div_t_z_3 = -J[2].y * t_z_inv;

	dJ_dmean[0].x = -h_x_div_t_z_2 * view_matrix[2];
	dJ_dmean[0].y = -h_x_div_t_z_2 * view_matrix[6];
	dJ_dmean[0].z = -h_x_div_t_z_2 * view_matrix[10];

	dJ_dmean[1].x = -h_x_div_t_z_2 * view_matrix[0] + 2.0f * h_x_t_x_div_t_z_3 * view_matrix[2];
	dJ_dmean[1].y = -h_x_div_t_z_2 * view_matrix[4] + 2.0f * h_x_t_x_div_t_z_3 * view_matrix[6];
	dJ_dmean[1].z = -h_x_div_t_z_2 * view_matrix[8] + 2.0f * h_x_t_x_div_t_z_3 * view_matrix[10];

	dJ_dmean[2].x = -h_y_div_t_z_2 * view_matrix[2];
	dJ_dmean[2].y = -h_y_div_t_z_2 * view_matrix[6];
	dJ_dmean[2].z = -h_y_div_t_z_2 * view_matrix[10];

	dJ_dmean[3].x = -h_y_div_t_z_2 * view_matrix[1] + 2.0f * h_y_t_y_div_t_z_3 * view_matrix[2];
	dJ_dmean[3].y = -h_y_div_t_z_2 * view_matrix[5] + 2.0f * h_y_t_y_div_t_z_3 * view_matrix[6];
	dJ_dmean[3].z = -h_y_div_t_z_2 * view_matrix[9] + 2.0f * h_y_t_y_div_t_z_3 * view_matrix[10];



	float8x3 dJ_dmean_2;

	
	const float pre_c1 = 2.0f * t_z_3_inv;
	const float pre_c2 = 4.0f * t_z_3_inv;
	const float pre_c3 = 6.0f * t_z_4_inv;

	const float v2_sq = view_matrix[2] * view_matrix[2];
	const float v6_sq = view_matrix[6] * view_matrix[6];
	const float v10_sq = view_matrix[10] * view_matrix[10];
	const float v2_v6 = view_matrix[2] * view_matrix[6];
	const float v10_v2 = view_matrix[10] * view_matrix[2];
	const float v10_v6 = view_matrix[10] * view_matrix[6];

	const float pre_factor_a = pre_c1 * h_x;
	const float pre_factor_e = pre_c1 * h_y;
	const float pre_factor_c_1 = pre_factor_a;
	const float pre_factor_c_2 = pre_c2 * h_x;
	const float pre_factor_c_3 = pre_c3 * h_x * t.x;
	const float pre_factor_f_1 = pre_factor_e;
	const float pre_factor_f_2 = pre_c2 * h_y;
	const float pre_factor_f_3 = pre_c3 * h_y * t.y;

	dJ_dmean_2[0].x = pre_factor_a * v2_sq;
	dJ_dmean_2[0].y = pre_factor_a * v2_v6;
	dJ_dmean_2[0].z = pre_factor_a * v10_v2;
	dJ_dmean_2[1].x = pre_factor_a * v6_sq;
	dJ_dmean_2[1].y = pre_factor_a * v10_v6;
	dJ_dmean_2[1].z = pre_factor_a * v10_sq;

	dJ_dmean_2[2].x = pre_factor_c_2 * view_matrix[0] * view_matrix[2] - pre_factor_c_3 * v2_sq;
	dJ_dmean_2[2].y = -pre_factor_c_3 * v2_v6 + pre_factor_c_1 * (view_matrix[0] * view_matrix[6] + view_matrix[2] * view_matrix[4]);
	dJ_dmean_2[2].z = -pre_factor_c_3 * v10_v2 + pre_factor_c_1 * (view_matrix[0] * view_matrix[10] + view_matrix[2] * view_matrix[8]);
	dJ_dmean_2[3].x = pre_factor_c_2 * view_matrix[4] * view_matrix[6] - pre_factor_c_3 * v6_sq;
	dJ_dmean_2[3].y = -pre_factor_c_3 * v10_v6 + pre_factor_c_1 * (view_matrix[10] * view_matrix[4] + view_matrix[6] * view_matrix[8]);
	dJ_dmean_2[3].x = pre_factor_c_2 * view_matrix[10] * view_matrix[8] - pre_factor_c_3 * v10_sq;

	dJ_dmean_2[4].x = pre_factor_e * v2_sq;
	dJ_dmean_2[4].y = pre_factor_e * v2_v6;
	dJ_dmean_2[4].z = pre_factor_e * v10_v2;
	dJ_dmean_2[5].x = pre_factor_e * v6_sq;
	dJ_dmean_2[5].y = pre_factor_e * v10_v6;
	dJ_dmean_2[5].z = pre_factor_e * v10_sq;

	dJ_dmean_2[6].x = pre_factor_f_2 * view_matrix[1] * view_matrix[2] - pre_factor_f_3 * v2_sq;
	dJ_dmean_2[6].y = -pre_factor_f_3 * v2_v6 + pre_factor_f_1 * (view_matrix[1] * view_matrix[6] + view_matrix[2] * view_matrix[5]);
	dJ_dmean_2[6].z = -pre_factor_f_3 * v10_v2 + pre_factor_f_1 * (view_matrix[1] * view_matrix[10] + view_matrix[2] * view_matrix[9]);
	dJ_dmean_2[7].x = pre_factor_f_2 * view_matrix[5] * view_matrix[6] - pre_factor_f_3 * v6_sq;
	dJ_dmean_2[7].y = -pre_factor_f_3 * v10_v6 + pre_factor_f_1 * (view_matrix[10] * view_matrix[5] + view_matrix[6] * view_matrix[9]);
	dJ_dmean_2[7].z = pre_factor_f_2 * view_matrix[10] * view_matrix[9] - pre_factor_f_3 * v10_sq;





	float3x3 dcov2D_dmean;

	dcov2D_dmean[0].x = dJ_dmean[0].x * term2[0].x + dJ_dmean[1].x * term2[0].z + dJ_dmean[0].x * term3[0].x + dJ_dmean[1].x * term3[2].x;
	dcov2D_dmean[0].y = dJ_dmean[0].y * term2[0].x + dJ_dmean[1].y * term2[0].z + dJ_dmean[0].y * term3[0].x + dJ_dmean[1].y * term3[2].x;
	dcov2D_dmean[0].z = dJ_dmean[0].z * term2[0].x + dJ_dmean[1].z * term2[0].z + dJ_dmean[0].z * term3[0].x + dJ_dmean[1].z * term3[2].x;

	dcov2D_dmean[1].x = dJ_dmean[0].x * term2[1].x + dJ_dmean[1].x * term2[1].z + dJ_dmean[2].x * term3[1].x + dJ_dmean[3].x * term3[2].x;
	dcov2D_dmean[1].y = dJ_dmean[0].y * term2[1].x + dJ_dmean[1].y * term2[1].z + dJ_dmean[2].y * term3[1].x + dJ_dmean[3].y * term3[2].x;
	dcov2D_dmean[1].z = dJ_dmean[0].z * term2[1].x + dJ_dmean[1].z * term2[1].z + dJ_dmean[2].z * term3[1].x + dJ_dmean[3].z * term3[2].x;

	dcov2D_dmean[2].x = dJ_dmean[2].x * term2[1].y + dJ_dmean[3].x * term2[1].z + dJ_dmean[2].x * term3[1].y + dJ_dmean[3].x * term3[2].y;
	dcov2D_dmean[2].y = dJ_dmean[2].y * term2[1].y + dJ_dmean[3].y * term2[1].z + dJ_dmean[2].y * term3[1].y + dJ_dmean[3].y * term3[2].y;
	dcov2D_dmean[2].z = dJ_dmean[2].z * term2[1].y + dJ_dmean[3].z * term2[1].z + dJ_dmean[2].z * term3[1].y + dJ_dmean[3].z * term3[2].y;



	float6x3 dcov2D_dmean_2;


	pre_t1 = term2[0].x + term3[0].x;
	pre_t2 = term2[0].z + term3[2].x;
	pre_t3 = term2[1].y + term3[1].y;
	pre_t4 = term2[1].z + term3[2].y;

	dcov2D_dmean_2[0].x = 2.0f * dJ_dmean[0].x * dJ_dmean[0].x * term1[0].x 
		+ 4.0f * dJ_dmean[0].x * dJ_dmean[1].x * term1[2].x
		+ 2.0f * dJ_dmean[1].x * dJ_dmean[1].x * term1[2].z
		+ dJ_dmean_2[0].x * pre_t1 + dJ_dmean_2[2].x * pre_t2;

	dcov2D_dmean_2[0].y = 2.0f * dJ_dmean[0].x * dJ_dmean[0].y * term1[0].x
		+ 2.0f * dJ_dmean[0].x * dJ_dmean[1].y * term1[2].x
		+ 2.0f * dJ_dmean[0].y * dJ_dmean[1].x * term1[2].x
		+ 2.0f * dJ_dmean[1].x * dJ_dmean[1].y * term1[2].z
		+ dJ_dmean_2[0].y * pre_t1 + dJ_dmean_2[2].y * pre_t2;

	dcov2D_dmean_2[0].z = 2.0f * dJ_dmean[0].x * dJ_dmean[0].z * term1[0].x
		+ 2.0f * dJ_dmean[0].x * dJ_dmean[1].z * term1[2].x
		+ 2.0f * dJ_dmean[0].z * dJ_dmean[1].x * term1[2].x
		+ 2.0f * dJ_dmean[1].x * dJ_dmean[1].z * term1[2].z
		+ dJ_dmean_2[0].z * pre_t1 + dJ_dmean_2[2].z * pre_t2;

	dcov2D_dmean_2[1].x = 2.0f * dJ_dmean[0].y * dJ_dmean[0].y * term1[0].x
		+ 4.0f * dJ_dmean[0].y * dJ_dmean[1].y * term1[2].x
		+ 2.0f * dJ_dmean[1].y * dJ_dmean[1].y * term1[2].z
		+ dJ_dmean_2[1].x * pre_t1 + dJ_dmean_2[3].x * pre_t2;

	dcov2D_dmean_2[1].y = 2.0f * dJ_dmean[0].y * dJ_dmean[0].z * term1[0].x
		+ 2.0f * dJ_dmean[0].y * dJ_dmean[1].z * term1[2].x
		+ 2.0f * dJ_dmean[0].z * dJ_dmean[1].y * term1[2].x
		+ 2.0f * dJ_dmean[1].y * dJ_dmean[1].z * term1[2].z
		+ dJ_dmean_2[1].y * pre_t1 + dJ_dmean_2[3].y * pre_t2;
	
	dcov2D_dmean_2[1].z = 2.0f * dJ_dmean[0].z * dJ_dmean[0].z * term1[0].x
		+ 4.0f * dJ_dmean[0].z * dJ_dmean[1].z * term1[2].x
		+ 2.0f * dJ_dmean[1].z * dJ_dmean[1].z * term1[2].z
		+ dJ_dmean_2[1].z * pre_t1 + dJ_dmean_2[3].z * pre_t2;
	


	dcov2D_dmean_2[2].x = 2.0f * dJ_dmean[0].x * dJ_dmean[2].x * term1[1].x
		+ 2.0f * dJ_dmean[0].x * dJ_dmean[3].x * term1[2].x
		+ 2.0f * dJ_dmean[1].x * dJ_dmean[2].x * term1[2].y
		+ 2.0f * dJ_dmean[1].x * dJ_dmean[3].x * term1[2].z
		+ dJ_dmean_2[0].x * term2[1].x
		+ dJ_dmean_2[2].x * term2[1].z
		+ dJ_dmean_2[4].x * term3[1].x
		+ dJ_dmean_2[6].x * term3[2].x;

	dcov2D_dmean_2[2].y = dJ_dmean[0].x * dJ_dmean[2].y * term1[1].x
		+ dJ_dmean[0].x * dJ_dmean[3].y * term1[2].x
		+ dJ_dmean[0].y * dJ_dmean[2].x * term1[1].x
		+ dJ_dmean[0].y * dJ_dmean[3].x * term1[2].x
		+ dJ_dmean[1].x * dJ_dmean[2].y * term1[2].y
		+ dJ_dmean[1].x * dJ_dmean[3].y * term1[2].z
		+ dJ_dmean[1].y * dJ_dmean[2].x * term1[2].y
		+ dJ_dmean[1].y * dJ_dmean[3].x * term1[2].z
		+ dJ_dmean_2[0].y * term2[1].x
		+ dJ_dmean_2[2].y * term2[1].z
		+ dJ_dmean_2[4].y * term3[1].x
		+ dJ_dmean_2[6].y * term3[2].x;

	dcov2D_dmean_2[2].z = dJ_dmean[0].x * dJ_dmean[2].z * term1[1].x
		+ dJ_dmean[0].x * dJ_dmean[3].z * term1[2].x
		+ dJ_dmean[0].z * dJ_dmean[2].x * term1[1].x
		+ dJ_dmean[0].z * dJ_dmean[3].x * term1[2].x
		+ dJ_dmean[1].x * dJ_dmean[2].z * term1[2].y
		+ dJ_dmean[1].x * dJ_dmean[3].z * term1[2].z
		+ dJ_dmean[1].z * dJ_dmean[2].x * term1[2].y
		+ dJ_dmean[1].z * dJ_dmean[3].x * term1[2].z
		+ dJ_dmean_2[0].z * term2[1].x
		+ dJ_dmean_2[2].z * term2[1].z
		+ dJ_dmean_2[4].z * term3[1].x
		+ dJ_dmean_2[6].z * term3[2].x;

	dcov2D_dmean_2[3].x = 2.0f * dJ_dmean[0].y * dJ_dmean[2].y * term1[1].x
		+ 2.0f * dJ_dmean[0].y * dJ_dmean[3].y * term1[2].x
		+ 2.0f * dJ_dmean[1].y * dJ_dmean[2].y * term1[2].y
		+ 2.0f * dJ_dmean[1].y * dJ_dmean[3].y * term1[2].z
		+ dJ_dmean_2[1].x * term2[1].x
		+ dJ_dmean_2[3].x * term2[1].z
		+ dJ_dmean_2[5].x * term3[1].x
		+ dJ_dmean_2[7].x * term3[2].x;

	dcov2D_dmean_2[3].y = dJ_dmean[0].y * dJ_dmean[2].z * term1[1].x
		+ dJ_dmean[0].y * dJ_dmean[3].z * term1[2].x
		+ dJ_dmean[0].z * dJ_dmean[2].y * term1[1].x
		+ dJ_dmean[0].z * dJ_dmean[3].y * term1[2].x
		+ dJ_dmean[1].y * dJ_dmean[2].z * term1[2].y
		+ dJ_dmean[1].y * dJ_dmean[3].z * term1[2].z
		+ dJ_dmean[1].z * dJ_dmean[2].y * term1[2].y
		+ dJ_dmean[1].z * dJ_dmean[3].y * term1[2].z
		+ dJ_dmean_2[1].y * term2[1].x
		+ dJ_dmean_2[3].y * term2[1].z
		+ dJ_dmean_2[5].y * term3[1].x
		+ dJ_dmean_2[7].y * term3[2].x;

	dcov2D_dmean_2[3].z = 2.0f * dJ_dmean[0].z * dJ_dmean[2].z * term1[1].x
		+ 2.0f * dJ_dmean[0].z * dJ_dmean[3].z * term1[2].x
		+ 2.0f * dJ_dmean[1].z * dJ_dmean[2].z * term1[2].y
		+ 2.0f * dJ_dmean[1].z * dJ_dmean[3].z * term1[2].z
		+ dJ_dmean_2[1].z * term2[1].x
		+ dJ_dmean_2[3].z * term2[1].z
		+ dJ_dmean_2[5].z * term3[1].x
		+ dJ_dmean_2[7].z * term3[2].x;




	dcov2D_dmean_2[4].x = 2.0f * dJ_dmean[2].x * dJ_dmean[2].x * term1[1].y
		+ 4.0f * dJ_dmean[2].x * dJ_dmean[3].x * term1[2].y
		+ 2.0f * dJ_dmean[3].x * dJ_dmean[3].x * term1[2].z
		+ dJ_dmean_2[4].x * pre_t3 + dJ_dmean_2[6].x * pre_t4;

	dcov2D_dmean_2[4].y = 2.0f * dJ_dmean[2].x * dJ_dmean[2].y * term1[1].y
		+ 2.0f * dJ_dmean[2].x * dJ_dmean[3].y * term1[2].y
		+ 2.0f * dJ_dmean[2].y * dJ_dmean[3].x * term1[2].y
		+ 2.0f * dJ_dmean[3].x * dJ_dmean[3].y * term1[2].z
		+ dJ_dmean_2[4].y * pre_t3 + dJ_dmean_2[6].y * pre_t4;

	dcov2D_dmean_2[4].z = 2.0f * dJ_dmean[2].x * dJ_dmean[2].z * term1[1].y
		+ 2.0f * dJ_dmean[2].x * dJ_dmean[3].z * term1[2].y
		+ 2.0f * dJ_dmean[2].z * dJ_dmean[3].x * term1[2].y
		+ 2.0f * dJ_dmean[3].x * dJ_dmean[3].z * term1[2].z
		+ dJ_dmean_2[4].z * pre_t3 + dJ_dmean_2[6].z * pre_t4;

	dcov2D_dmean_2[5].x = 2.0f * dJ_dmean[2].y * dJ_dmean[2].y * term1[1].y
		+ 4.0f * dJ_dmean[2].y * dJ_dmean[3].y * term1[2].y
		+ 2.0f * dJ_dmean[3].y * dJ_dmean[3].y * term1[2].z
		+ dJ_dmean_2[5].x * pre_t3 + dJ_dmean_2[7].x * pre_t4;

	dcov2D_dmean_2[5].y = 2.0f * dJ_dmean[2].y * dJ_dmean[2].z * term1[1].y
		+ 2.0f * dJ_dmean[2].y * dJ_dmean[3].z * term1[2].y
		+ 2.0f * dJ_dmean[2].z * dJ_dmean[3].y * term1[2].y
		+ 2.0f * dJ_dmean[3].y * dJ_dmean[3].z * term1[2].z
		+ dJ_dmean_2[5].y * pre_t3 + dJ_dmean_2[7].y * pre_t4;

	dcov2D_dmean_2[5].z = 2.0f * dJ_dmean[2].z * dJ_dmean[2].z * term1[1].y
		+ 4.0f * dJ_dmean[2].z * dJ_dmean[3].z * term1[2].y
		+ 2.0f * dJ_dmean[3].z * dJ_dmean[3].z * term1[2].z
		+ dJ_dmean_2[5].z * pre_t3 + dJ_dmean_2[7].z * pre_t4;




	dG_dmean.x += dcov2D_dmean[0].x * dG_dc_xx + 2.0f * dcov2D_dmean[1].x * dG_dc_xy + dcov2D_dmean[2].x * dG_dc_yy;
	dG_dmean.y += dcov2D_dmean[0].y * dG_dc_xx + 2.0f * dcov2D_dmean[1].y * dG_dc_xy + dcov2D_dmean[2].y * dG_dc_yy;
	dG_dmean.z += dcov2D_dmean[0].z * dG_dc_xx + 2.0f * dcov2D_dmean[1].z * dG_dc_xy + dcov2D_dmean[2].z * dG_dc_yy;


	dG_dmean_2[0].x += dG_dc_xx * dcov2D_dmean_2[0].x + 2.0f * dG_dc_xy * dcov2D_dmean_2[2].x + dG_dc_yy * dcov2D_dmean_2[4].x;
	dG_dmean_2[0].y += dG_dc_xx * dcov2D_dmean_2[0].y + 2.0f * dG_dc_xy * dcov2D_dmean_2[2].y + dG_dc_yy * dcov2D_dmean_2[4].y;
	dG_dmean_2[0].z += dG_dc_xx * dcov2D_dmean_2[0].z + 2.0f * dG_dc_xy * dcov2D_dmean_2[2].z + dG_dc_yy * dcov2D_dmean_2[4].z;

	dG_dmean_2[1].x += dG_dc_xx * dcov2D_dmean_2[1].x + 2.0f * dG_dc_xy * dcov2D_dmean_2[3].x + dG_dc_yy * dcov2D_dmean_2[5].x;
	dG_dmean_2[1].y += dG_dc_xx * dcov2D_dmean_2[1].y + 2.0f * dG_dc_xy * dcov2D_dmean_2[3].y + dG_dc_yy * dcov2D_dmean_2[5].y;
	dG_dmean_2[1].z += dG_dc_xx * dcov2D_dmean_2[1].z + 2.0f * dG_dc_xy * dcov2D_dmean_2[3].z + dG_dc_yy * dcov2D_dmean_2[5].z;





	dG_dmean_2[0].x += dcov2D_dmean[0].x * dcov2D_dmean[0].x * dG_dcov2D_2[0].x +
		2.0f * dcov2D_dmean[0].x * dcov2D_dmean[1].x * (dG_dcov2D_2[0].y + dG_dcov2D_2[1].x) + 
		2.0f * dcov2D_dmean[1].x * dcov2D_dmean[1].x * dG_dcov2D_2[1].y + 
		dcov2D_dmean[0].x * dcov2D_dmean[2].x * (dG_dcov2D_2[0].z + dG_dcov2D_2[2].x) + 
		2.0f * dcov2D_dmean[1].x * dcov2D_dmean[2].x * (dG_dcov2D_2[1].z + dG_dcov2D_2[2].y) + 
		dcov2D_dmean[2].x * dcov2D_dmean[2].x * dG_dcov2D_2[2].z;

	dG_dmean_2[0].y += dcov2D_dmean[0].x * (dcov2D_dmean[0].y * dG_dcov2D_2[0].x + 2.0f * dcov2D_dmean[1].y * dG_dcov2D_2[0].y + dcov2D_dmean[2].y * dG_dcov2D_2[0].z) +
		2.0f * dcov2D_dmean[0].y * dcov2D_dmean[1].x * dG_dcov2D_2[1].x +
		dcov2D_dmean[0].y * dcov2D_dmean[2].x * dG_dcov2D_2[2].x +
		2.0f * dcov2D_dmean[1].x * dcov2D_dmean[1].y * dG_dcov2D_2[1].y +
		2.0f * dcov2D_dmean[1].x * dcov2D_dmean[2].y * dG_dcov2D_2[1].z +
		2.0f * dcov2D_dmean[1].y * dcov2D_dmean[2].x * dG_dcov2D_2[2].y +
		dcov2D_dmean[2].x * dcov2D_dmean[2].y * dG_dcov2D_2[2].z;

	dG_dmean_2[0].z += dcov2D_dmean[0].x * (dcov2D_dmean[0].z * dG_dcov2D_2[0].x + 2.0f * dcov2D_dmean[1].z * dG_dcov2D_2[0].y + dcov2D_dmean[2].z * dG_dcov2D_2[0].z) +
		2.0f * dcov2D_dmean[0].z * dcov2D_dmean[1].x * dG_dcov2D_2[1].x +
		dcov2D_dmean[0].z * dcov2D_dmean[2].x * dG_dcov2D_2[2].x +
		2.0f * dcov2D_dmean[1].x * dcov2D_dmean[1].z * dG_dcov2D_2[1].y +
		2.0f * dcov2D_dmean[1].x * dcov2D_dmean[2].z * dG_dcov2D_2[1].z +
		2.0f * dcov2D_dmean[1].z * dcov2D_dmean[2].x * dG_dcov2D_2[2].y +
		dcov2D_dmean[2].x * dcov2D_dmean[2].z * dG_dcov2D_2[2].z;


	dG_dmean_2[1].x += dcov2D_dmean[0].y * dcov2D_dmean[0].y * dG_dcov2D_2[0].x +
		2.0f * dcov2D_dmean[0].y * dcov2D_dmean[1].y * (dG_dcov2D_2[0].y + dG_dcov2D_2[1].x) +
		2.0f * dcov2D_dmean[1].y * dcov2D_dmean[1].y * dG_dcov2D_2[1].y +
		dcov2D_dmean[0].y * dcov2D_dmean[2].y * (dG_dcov2D_2[0].z + dG_dcov2D_2[2].x) +
		2.0f * dcov2D_dmean[1].y * dcov2D_dmean[2].y * (dG_dcov2D_2[1].z + dG_dcov2D_2[2].y) +
		dcov2D_dmean[2].y * dcov2D_dmean[2].y * dG_dcov2D_2[2].z;

	dG_dmean_2[1].y += dcov2D_dmean[0].y * (dcov2D_dmean[0].z * dG_dcov2D_2[0].x + 2.0f * dcov2D_dmean[1].z * dG_dcov2D_2[0].y + dcov2D_dmean[2].z * dG_dcov2D_2[0].z) +
		2.0f * dcov2D_dmean[0].z * dcov2D_dmean[1].y * dG_dcov2D_2[1].x +
		dcov2D_dmean[0].z * dcov2D_dmean[2].y * dG_dcov2D_2[2].x +
		2.0f * dcov2D_dmean[1].y * dcov2D_dmean[1].z * dG_dcov2D_2[1].y +
		2.0f * dcov2D_dmean[1].y * dcov2D_dmean[2].z * dG_dcov2D_2[1].z +
		2.0f * dcov2D_dmean[1].z * dcov2D_dmean[2].y * dG_dcov2D_2[2].y +
		dcov2D_dmean[2].y * dcov2D_dmean[2].z * dG_dcov2D_2[2].z;

	dG_dmean_2[1].z += dcov2D_dmean[0].z * dcov2D_dmean[0].z * dG_dcov2D_2[0].x +
		2.0f * dcov2D_dmean[0].z * dcov2D_dmean[1].z * (dG_dcov2D_2[0].y + dG_dcov2D_2[1].x) +
		2.0f * dcov2D_dmean[1].z * dcov2D_dmean[1].z * dG_dcov2D_2[1].y +
		dcov2D_dmean[0].z * dcov2D_dmean[2].z * (dG_dcov2D_2[0].z + dG_dcov2D_2[2].x) +
		2.0f * dcov2D_dmean[1].z * dcov2D_dmean[2].z * (dG_dcov2D_2[1].z + dG_dcov2D_2[2].y) +
		dcov2D_dmean[2].z * dcov2D_dmean[2].z * dG_dcov2D_2[2].z;


	

	float3x2 dG_dcov2D_dmean2D;

	dG_dcov2D_dmean2D[0].x = (
		dG_dconic2D_dmean2D[0].x * dconic2D_dcov2D[0].x
		+ 2.0f * dG_dconic2D_dmean2D[1].x * dconic2D_dcov2D[1].x
		+ dG_dconic2D_dmean2D[2].x * dconic2D_dcov2D[2].x
	);
	dG_dcov2D_dmean2D[0].y = (
		dG_dconic2D_dmean2D[0].y * dconic2D_dcov2D[0].x
		+ 2.0f * dG_dconic2D_dmean2D[1].y * dconic2D_dcov2D[1].x
		+ dG_dconic2D_dmean2D[2].y * dconic2D_dcov2D[2].x
	);

	dG_dcov2D_dmean2D[1].x = (
		dG_dconic2D_dmean2D[0].x * dconic2D_dcov2D[0].y
		+ dG_dconic2D_dmean2D[1].x * dconic2D_dcov2D[1].y
		+ dG_dconic2D_dmean2D[2].x * dconic2D_dcov2D[2].y
	);
	dG_dcov2D_dmean2D[1].y = (
		dG_dconic2D_dmean2D[0].y * dconic2D_dcov2D[0].y
		+ dG_dconic2D_dmean2D[1].y * dconic2D_dcov2D[1].y
		+ dG_dconic2D_dmean2D[2].y * dconic2D_dcov2D[2].y
	);
		
	dG_dcov2D_dmean2D[2].x = (
		dG_dconic2D_dmean2D[0].x * dconic2D_dcov2D[0].z
		+ 2.0f * dG_dconic2D_dmean2D[1].x * dconic2D_dcov2D[1].z
		+ dG_dconic2D_dmean2D[2].x * dconic2D_dcov2D[2].z
	);
	dG_dcov2D_dmean2D[2].y = (
		dG_dconic2D_dmean2D[0].y * dconic2D_dcov2D[0].z
		+ 2.0f * dG_dconic2D_dmean2D[1].y * dconic2D_dcov2D[1].z
		+ dG_dconic2D_dmean2D[2].y * dconic2D_dcov2D[2].z
	);





	float2x3 dG_dmean2D_dcov2D;

	dG_dmean2D_dcov2D[0].x = (
		dconic2D_dcov2D[0].x * dG_dmean2D_dconic2D[0].x 
		+ 2.0f * dconic2D_dcov2D[1].x * dG_dmean2D_dconic2D[0].y 
		+ dconic2D_dcov2D[2].x * dG_dmean2D_dconic2D[0].z
	);
	dG_dmean2D_dcov2D[0].y = (
		dconic2D_dcov2D[0].y * dG_dmean2D_dconic2D[0].x 
		+ 2.0f * dconic2D_dcov2D[1].y * dG_dmean2D_dconic2D[0].y 
		+ dconic2D_dcov2D[2].y * dG_dmean2D_dconic2D[0].z
	);
	dG_dmean2D_dcov2D[0].z = (
		dconic2D_dcov2D[0].z * dG_dmean2D_dconic2D[0].x 
		+ 2.0f * dconic2D_dcov2D[1].z * dG_dmean2D_dconic2D[0].y 
		+ dconic2D_dcov2D[2].z * dG_dmean2D_dconic2D[0].z
	);

	dG_dmean2D_dcov2D[1].x = (
		dconic2D_dcov2D[0].x * dG_dmean2D_dconic2D[1].x 
		+ 2.0f * dconic2D_dcov2D[1].x * dG_dmean2D_dconic2D[1].y 
		+ dconic2D_dcov2D[2].x * dG_dmean2D_dconic2D[1].z
	);
	dG_dmean2D_dcov2D[1].y = (
		dconic2D_dcov2D[0].y * dG_dmean2D_dconic2D[1].x 
		+ 2.0f * dconic2D_dcov2D[1].y * dG_dmean2D_dconic2D[1].y 
		+ dconic2D_dcov2D[2].y * dG_dmean2D_dconic2D[1].z
	);
	dG_dmean2D_dcov2D[1].z = (
		dconic2D_dcov2D[0].z * dG_dmean2D_dconic2D[1].x 
		+ 2.0f * dconic2D_dcov2D[1].z * dG_dmean2D_dconic2D[1].y 
		+ dconic2D_dcov2D[2].z * dG_dmean2D_dconic2D[1].z
	);





	const float term_1 = dmean2D_dmean3D[0].z * dG_dcov2D_dmean2D[0].x + dmean2D_dmean3D[1].z * dG_dcov2D_dmean2D[0].y;
	const float term_2 = dmean2D_dmean3D[0].z * dG_dcov2D_dmean2D[1].x + dmean2D_dmean3D[1].z * dG_dcov2D_dmean2D[1].y;
	const float term_3 = dmean2D_dmean3D[0].z * dG_dcov2D_dmean2D[2].x + dmean2D_dmean3D[1].z * dG_dcov2D_dmean2D[2].y;
	const float term_4 = dmean2D_dmean3D[0].y * dG_dcov2D_dmean2D[0].x + dmean2D_dmean3D[1].y * dG_dcov2D_dmean2D[0].y;
	const float term_5 = dmean2D_dmean3D[0].y * dG_dcov2D_dmean2D[1].x + dmean2D_dmean3D[1].y * dG_dcov2D_dmean2D[1].y;
	const float term_6 = dmean2D_dmean3D[0].y * dG_dcov2D_dmean2D[2].x + dmean2D_dmean3D[1].y * dG_dcov2D_dmean2D[2].y;

	dG_dmean_2[0].x += (
		dcov2D_dmean[0].x * (dmean2D_dmean3D[0].x * dG_dcov2D_dmean2D[0].x + dmean2D_dmean3D[1].x * dG_dcov2D_dmean2D[0].y)
		+ 2.0f * dcov2D_dmean[1].x * (dmean2D_dmean3D[0].x * dG_dcov2D_dmean2D[1].x + dmean2D_dmean3D[1].x * dG_dcov2D_dmean2D[1].y)
		+ dcov2D_dmean[2].x * (dmean2D_dmean3D[0].x * dG_dcov2D_dmean2D[2].x + dmean2D_dmean3D[1].x * dG_dcov2D_dmean2D[2].y)
	);
	dG_dmean_2[0].y += (
		dcov2D_dmean[0].x * (term_4)
		+ 2.0f * dcov2D_dmean[1].x * (term_5)
		+ dcov2D_dmean[2].x * (term_6)
	);

	dG_dmean_2[0].z += (
		dcov2D_dmean[0].x * (term_1)
		+ 2.0f * dcov2D_dmean[1].x * (term_2)
		+ dcov2D_dmean[2].x * (term_3)
	);

	dG_dmean_2[1].x += (
		dcov2D_dmean[0].y * (term_4)
		+ 2.0f * dcov2D_dmean[1].y * (term_5)
		+ dcov2D_dmean[2].y * (term_6)
	);
	dG_dmean_2[1].y += (
		dcov2D_dmean[0].y * (term_1)
		+ 2.0f * dcov2D_dmean[1].y * (term_2)
		+ dcov2D_dmean[2].y * (term_3)
	);
	dG_dmean_2[1].z += (
		dcov2D_dmean[0].z * (term_1)
		+ 2.0f * dcov2D_dmean[1].z * (term_2)
		+ dcov2D_dmean[2].z * (term_3)
	);

	const float term_7 = dmean2D_dmean3D[0].x * dG_dmean2D_dcov2D[0].x + dmean2D_dmean3D[1].x * dG_dmean2D_dcov2D[1].x;
	const float term_8 = dmean2D_dmean3D[0].x * dG_dmean2D_dcov2D[0].y + dmean2D_dmean3D[1].x * dG_dmean2D_dcov2D[1].y;
	const float term_9 = dmean2D_dmean3D[0].x * dG_dmean2D_dcov2D[0].z + dmean2D_dmean3D[1].x * dG_dmean2D_dcov2D[1].z;
	const float term_10 = dmean2D_dmean3D[0].y * dG_dmean2D_dcov2D[0].x + dmean2D_dmean3D[1].y * dG_dmean2D_dcov2D[1].x;
	const float term_11 = dmean2D_dmean3D[0].y * dG_dmean2D_dcov2D[0].y + dmean2D_dmean3D[1].y * dG_dmean2D_dcov2D[1].y;
	const float term_12 = dmean2D_dmean3D[0].y * dG_dmean2D_dcov2D[0].z + dmean2D_dmean3D[1].y * dG_dmean2D_dcov2D[1].z;



	dG_dmean_2[0].x += (
		dcov2D_dmean[0].x * (term_7)
		+ 2.0f * dcov2D_dmean[1].x * (term_8)
		+ dcov2D_dmean[2].x * (term_9)
	);
	dG_dmean_2[0].y += (
		dcov2D_dmean[0].y * (term_7)
		+ 2.0f * dcov2D_dmean[1].y * (term_8)
		+ dcov2D_dmean[2].y * (term_9)
	);
	dG_dmean_2[0].z += (
		dcov2D_dmean[0].z * (term_7)
		+ 2.0f * dcov2D_dmean[1].z * (term_8)
		+ dcov2D_dmean[2].z * (term_9)
	);

	dG_dmean_2[1].x += (
		dcov2D_dmean[0].y * (term_10)
		+ 2.0f * dcov2D_dmean[1].y * (term_11)
		+ dcov2D_dmean[2].y * (term_12)
	);

	dG_dmean_2[1].y += (
		dcov2D_dmean[0].z * (term_10)
		+ 2.0f * dcov2D_dmean[1].z * (term_11)
		+ dcov2D_dmean[2].z * (term_12)
	);

	dG_dmean_2[1].z += (
		dcov2D_dmean[0].z * (dmean2D_dmean3D[0].z * dG_dmean2D_dcov2D[0].x + dmean2D_dmean3D[1].z * dG_dmean2D_dcov2D[1].x)
		+ 2.0f * dcov2D_dmean[1].z * (dmean2D_dmean3D[0].z * dG_dmean2D_dcov2D[0].y + dmean2D_dmean3D[1].z * dG_dmean2D_dcov2D[1].y)
		+ dcov2D_dmean[2].z * (dmean2D_dmean3D[0].z * dG_dmean2D_dcov2D[0].z + dmean2D_dmean3D[1].z * dG_dmean2D_dcov2D[1].z)
	);





// Backward version of the rendering procedure.
template <uint32_t C>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float* __restrict__ bg_color,
	const float2* __restrict__ points_xy_image,
	const float4* __restrict__ conic_opacity,
	const float* __restrict__ colors,
	const float* __restrict__ depths,
	const float* __restrict__ final_Ts,
	const uint32_t* __restrict__ n_contrib,



	float* __restrict__ dpixel_dcolors,
	float3* __restrict__ dpixel_dG,
	

	float2* __restrict__ dG_dmean2D,
	float3* __restrict__ dG_dmean2D_2,

	float3* __restrict__ dG_dconic2D,

	float3x3* __restrict__ dG_dconic2D_dcov2D,
	float2x3* __restrict__ dG_dmean2D_dconic2D,
	float3X2* __restrict__ dG_dconic2D_dmean2D,

	// for dG_dconic2D_2, a 2,2,2,2 hessian matrix looks like:
	// [ab][bd]
	// [bc][de]
	// [bd][ce]
	// [de][ef]

	// for dG_dmean2D_dconic2D, a 2*2*2 hessian matrix looks like:
	// [a][b]
	// [b][c]

	// [d][e]
	// [e][f]

	// for dG_dconic2D_dcov2D, a 2*2*2*2 hessian matrix looks like:
	// [a b][d e]
	// [b c][e f]
	
	// [d e][g h]
	// [e f][h i]

	float* __restrict__ dpixel_dopacity,

	// float3* __restrict__ dG_dconic2D_2_abc,
	// float3* __restrict__ dG_dconic2D_2_def,
)
{
	// We rasterize again. Compute necessary block info.
	auto block = cg::this_thread_block();

	//水平方向block个数
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;

	//block负责的像素块左下角像素的坐标
	const uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	//block负责的像素块右上角像素的坐标
	const uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };

	//thread负责的像素的坐标
	const uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	//thread负责的像素idx
	const uint32_t pix_id = W * pix.y + pix.x;

	//i * H * W + pix_id 代表 thread负责的像素在第i个通道的内存idx
	//block.thread_rank() 代表 thread在block中的局部idx

	const float2 pixf = { (float)pix.x, (float)pix.y };

	const bool inside = pix.x < W&& pix.y < H;
	const uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];

	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);

	bool done = !inside;
	int toDo = range.y - range.x;

	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];
	__shared__ float collected_colors[C * BLOCK_SIZE];
	__shared__ float collected_depths[BLOCK_SIZE];


	// In the forward, we stored the final value for T, the
	// product of all (1 - alpha) factors. 
	const float T_final = inside ? final_Ts[pix_id] : 0;
	float T = T_final;

	// We start from the back. The ID of the last contributing
	// Gaussian is known from each pixel from the forward.
	uint32_t contributor = toDo;
	const int last_contributor = inside ? n_contrib[pix_id] : 0;

	float accum_rec[C] = { 0 };

	float last_alpha = 0;
	float last_color[C] = { 0 };


	// Traverse all Gaussians
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// Load auxiliary data into shared memory, start in the BACK
		// and load them in revers order.
		block.sync();
		const int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			const int coll_id = point_list[range.y - progress - 1];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
			for (int i = 0; i < C; i++)
				collected_colors[i * BLOCK_SIZE + block.thread_rank()] = colors[coll_id * C + i];

			if(dL_invdepths)
			collected_depths[block.thread_rank()] = depths[coll_id];
		}
		block.sync();

		// Iterate over Gaussians
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current Gaussian ID. Skip, if this one
			// is behind the last contributor for this pixel.
			contributor--;
			if (contributor >= last_contributor)
				continue;

			const int global_id = collected_id[j];

			// Compute blending values, as before.
			const float2 xy = collected_xy[j];
			const float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			const float4 con_o = collected_conic_opacity[j];
			const float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			const float G = exp(power);
			const float alpha = min(0.99f, con_o.w * G);
			if (alpha < 1.0f / 255.0f)
				continue;

			T = T / (1.f - alpha);

			// Propagate gradients to per-Gaussian colors and keep
			// gradients w.r.t. alpha (blending factor for a Gaussian/pixel
			// pair).
			float3 dpixel_dalpha = 0.0f;
			float* dpixel_dalpha_ptr = (float*)&dpixel_dalpha;

			

			for (int ch = 0; ch < C; ch++)
			{
				const float c = collected_colors[ch * BLOCK_SIZE + j];
				// Update last color (to be used in the next iteration)
				accum_rec[ch] = last_alpha * last_color[ch] + (1.f - last_alpha) * accum_rec[ch];
				last_color[ch] = c;

				dpixel_dalpha_ptr[ch] = (c - accum_rec[ch]);
				// Update the gradients w.r.t. color of the Gaussian. 
				// Atomic, since this pixel is just one of potentially
				// many that were affected by this Gaussian.
			}
			atomicAdd(&(dpixel_dcolors[global_id]), alpha * T);
			dpixel_dalpha *= T;

			// Update last alpha (to be used in the next iteration)
			last_alpha = alpha;

			for (int i = 0; i < C; i++)
				dpixel_dalpha_ptr[i] += (-T_final / (1.f - alpha)) * bg_color[i];
			

			

			const float gdx = G * d.x;
			const float gdy = G * d.y;

			float* dpixel_dG_ptr = (float*)&dpixel_dG[global_id];
			for (int i = 0; i < C; ++i){
				dpixel_dG_ptr[i] = con_o.w * dpixel_dalpha_ptr[i]
			}
			

			const float3 dx_2 = {d.x * d.x, d.x * d.y, d.y * d.y};


			const float W_div_2 = 0.5f * W;
			const float H_div_2 = 0.5f * H;

			// Update gradients w.r.t. 2D mean position of the Gaussian
			atomicAdd(&dG_dmean2D[global_id].x, (-gdx * con_o.x - gdy * con_o.y) * W_div_2);
			atomicAdd(&dG_dmean2D[global_id].y, (-gdy * con_o.z - gdx * con_o.y) * H_div_2);


			const float conic_y_2 = con_o.y * con_o.y;
			const float term1 = (-1.f + 2.f * con_o.y * dx_2.y);
			atomicAdd(&dG_dmean2D_2[global_id].x, G * (con_o.x * con_o.x * dx_2.x + conic_y_2 * dx_2.z + con_o.x * term1) * W_div_2 * W_div_2);
			atomicAdd(&dG_dmean2D_2[global_id].y, G * (conic_y_2 * dx_2.y + con_o.x * con_o.z * dx_2.y + con_o.y * (-1.f + con_o.x * dx_2.x + con_o.z * dx_2.z)) * W_div_2 * H_div_2);
			atomicAdd(&dG_dmean2D_2[global_id].z, G * (conic_y_2 * dx_2.x + con_o.z * con_o.z * dx_2.z + con_o.z * term1) * H_div_2 * H_div_2);


			// Update gradients w.r.t. 2D covariance (2x2 matrix, symmetric)
			atomicAdd(&dG_dconic2D[global_id].x, -0.5f * gdx * d.x);
			atomicAdd(&dG_dconic2D[global_id].y, -0.5f * gdx * d.y);
			atomicAdd(&dG_dconic2D[global_id].z, -0.5f * gdy * d.y);
			
			const float term2 = 0.25f * G * dx_2.x;
			const float term3 = 0.25f * G * dx_2.z;
			
			// atomicAdd(&dG_dconic2D_2_abc[global_id].x, term2 * dx_2.x);
			// atomicAdd(&dG_dconic2D_2_abc[global_id].y, term2 * dx_2.y);
			// atomicAdd(&dG_dconic2D_2_abc[global_id].z, term2 * dx_2.z);
			// atomicAdd(&dG_dconic2D_2_def[global_id].x, term2 * dx_2.z);
			// atomicAdd(&dG_dconic2D_2_def[global_id].y, term3 * dx_2.y);
			// atomicAdd(&dG_dconic2D_2_def[global_id].z, term3 * dx_2.z);


			const float term4 = con_o.x * dx_2.x + con_o.y * dx_2.y;
			const float term5 = con_o.y * dx_2.y + con_o.z * dx_2.z;
			const float term6 = G * d.x * 0.5f;
			const float term7 = G * d.y * 0.5f;


			const float common_factor = -0.25f * exp(-0.5f * (term4 + term5));
			const float p1 = con_o.x * d.x + con_o.y * d.y;
			const float p2 = con_o.y * d.x + con_o.z * d.y;

			const float p1_sq = p1 * p1;
			const float p2_sq = p2 * p2;
			const float p1_p2 = p1 * p2;
		
			atomicAdd(&dG_dmean2D_dconic2D[global_id][0].x, term6 * (-2.f + term4));
			atomicAdd(&dG_dmean2D_dconic2D[global_id][0].y, term7 * (-1.f + term4));
			atomicAdd(&dG_dmean2D_dconic2D[global_id][0].z, term3 * 2.f * p1);
			atomicAdd(&dG_dmean2D_dconic2D[global_id][1].x, term2 * 2.f * p2);
			atomicAdd(&dG_dmean2D_dconic2D[global_id][1].y, term6 * (-1.f + term5));
			atomicAdd(&dG_dmean2D_dconic2D[global_id][1].z, term7 * (-2.f + term5));




			atomicAdd(dG_dconic2D_dcov2D[global_id][0].x, common_factor * p1_sq * dx_2.x);
			atomicAdd(dG_dconic2D_dcov2D[global_id][0].y, common_factor * p1_p2 * dx_2.x);
			atomicAdd(dG_dconic2D_dcov2D[global_id][0].z, common_factor * p2_sq * dx_2.x);

			atomicAdd(dG_dconic2D_dcov2D[global_id][1].x, common_factor * p1_sq * dx_2.y);
			atomicAdd(dG_dconic2D_dcov2D[global_id][1].y, common_factor * p1_p2 * dx_2.y);
			atomicAdd(dG_dconic2D_dcov2D[global_id][1].z, common_factor * p2_sq * dx_2.y);

			atomicAdd(dG_dconic2D_dcov2D[global_id][2].x, common_factor * p1_sq * dx_2.z);
			atomicAdd(dG_dconic2D_dcov2D[global_id][2].y, common_factor * p1_p2 * dx_2.z);
			atomicAdd(dG_dconic2D_dcov2D[global_id][2].z, common_factor * p2_sq * dx_2.z);



			atomicAdd(&dG_dconic2D_dmean2D[global_id][0].x, term6 * W_div_2 * (-2.f + term4));
			atomicAdd(&dG_dconic2D_dmean2D[global_id][0].y, term2 * H * p2);

			atomicAdd(&dG_dconic2D_dmean2D[global_id][1].x, term7 * W_div_2 * (-1.f + term4));
			atomicAdd(&dG_dconic2D_dmean2D[global_id][1].y, term6 * H_div_2 * (-1.f + term5));

			atomicAdd(&dG_dconic2D_dmean2D[global_id][2].x, term3 * W * p1);
			atomicAdd(&dG_dconic2D_dmean2D[global_id][2].y, term7 * H_div_2 * (-2.f + term5));




			// Update gradients w.r.t. opacity of the Gaussian
			atomicAdd(&(dL_dopacity[global_id]), G * dL_dalpha);
		}
	}
}

void BACKWARD::preprocess(
	int P, int D, int M,
	const float3* means3D,
	const int* radii,
	const float* shs,
	const bool* clamped,
	const float* opacities,
	const glm::vec3* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* cov3Ds,
	const float* viewmatrix,
	const float* projmatrix,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	const glm::vec3* campos,
	const float3* dL_dmean2D,
	const float* dL_dconic,
	const float* dL_dinvdepth,
	float* dL_dopacity,
	glm::vec3* dL_dmean3D,
	float* dL_dcolor,
	float* dL_dcov3D,
	float* dL_dsh,
	glm::vec3* dL_dscale,
	glm::vec4* dL_drot,
	bool antialiasing)
{
	// Propagate gradients for the path of 2D conic matrix computation. 
	// Somewhat long, thus it is its own kernel rather than being part of 
	// "preprocess". When done, loss gradient w.r.t. 3D means has been
	// modified and gradient w.r.t. 3D covariance matrix has been computed.	
	computeCov2DCUDA << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		radii,
		cov3Ds,
		focal_x,
		focal_y,
		tan_fovx,
		tan_fovy,
		viewmatrix,
		opacities,
		dL_dconic,
		dL_dopacity,
		dL_dinvdepth,
		(float3*)dL_dmean3D,
		dL_dcov3D,
		antialiasing);

	// Propagate gradients for remaining steps: finish 3D mean gradients,
	// propagate color gradients to SH (if desireD), propagate 3D covariance
	// matrix gradients to scale and rotation.
	preprocessCUDA<NUM_CHANNELS> << < (P + 255) / 256, 256 >> > (
		P, D, M,
		(float3*)means3D,
		radii,
		shs,
		clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		projmatrix,
		campos,
		(float3*)dL_dmean2D,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		dL_dscale,
		dL_drot,
		dL_dopacity);
}

void BACKWARD::render(
	const dim3 grid, const dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float* bg_color,
	const float2* means2D,
	const float4* conic_opacity,
	const float* colors,
	const float* depths,
	const float* final_Ts,
	const uint32_t* n_contrib,
	const float* dL_dpixels,
	const float* dL_invdepths,
	float3* dL_dmean2D,
	float4* dL_dconic2D,
	float* dL_dopacity,
	float* dL_dcolors,
	float* dL_dinvdepths)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> >(
		ranges,
		point_list,
		W, H,
		bg_color,
		means2D,
		conic_opacity,
		colors,
		depths,
		final_Ts,
		n_contrib,
		dL_dpixels,
		dL_invdepths,
		dL_dmean2D,
		dL_dconic2D,
		dL_dopacity,
		dL_dcolors,
		dL_dinvdepths
		);
}
